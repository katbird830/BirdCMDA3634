#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ unsigned int dProdMod(unsigned int a, unsigned int b, unsigned int p) {
	unsigned int za = a;
	unsigned int ab = 0;

	while (b>0) {
		if (b%2 == 1) ab = (ab+za)%p;
		za = (2*za)%p;
		b/=2;
	}
	return ab;
}

__device__ unsigned int dExpMod(unsigned int a, unsigned int b, unsigned int p) {
	unsigned int z = a;
	unsigned int aExpb = 1;

	while (b>0) {
		if (b%2 == 1) {
			aExpb = dProdMod(aExpb, z, p);
		}
		z = dProdMod(z, z, p);
		b /= 2;
	}
	return aExpb;
}

__global__ void kernalFindKey(unsigned int p, unsigned int g, unsigned int h, unsigned int *x) {

	unsigned int threadid = (unsigned int) threadIdx.x;
	unsigned int blockid = (unsigned int) blockIdx.x;
	unsigned int Nblock = (unsigned int) blockDim.x;

	unsigned int id = threadid + blockid*Nblock + 1;

	if (dExpMod(g, id, p) == h) {
		*x = id;
	}
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */
	  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
	FILE *f;
	f = fopen("public_key.txt", "r");
	fscanf(f, "%u\n%u\n%u\n%u", &n, &p, &g, &h);
	fclose(f);

	FILE *fr;
	fr = fopen("message.txt", "r");
	fscanf(fr, "%u\n", &Nints);

	//allocating memory for the (m, a)
	unsigned int *b = (unsigned int *) malloc(Nints*sizeof(unsigned int));
	unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));

	//filling b and Zmessage from txt file
	for (unsigned int i = 0; i<Nints; i++) {
		fscanf(fr, "(%u,%u)\n", &Zmessage[i], &b[i]);
	}
	fclose(fr);

	//Q4
	unsigned int *d_x;

	hipMalloc(&d_x,1*sizeof(unsigned int));

	int Nthreads = 32;
	int Nblocks = (p+Nthreads-1)/Nthreads;
	
	double startTime = clock();
  // find the secret key
	if (x==0 || modExp(g,x,p)!=h) {
    	printf("Finding the secret key...\n");
    	double startTime = clock();
    	kernalFindKey <<<Nblocks ,Nthreads >>>(p, g, h, d_x);
	}
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  
	hipMemcpy(&x, d_x,1*sizeof(unsigned int), hipMemcpyDeviceToHost);

	unsigned int charsPerInt = (n-1)/8;
	unsigned int Nchars = Nints*charsPerInt;

	int bufferSize = 1024;
	unsigned char *message1 = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));

	ElGamalDecrypt(Zmessage, b, Nints, p, x);
	
	convertZToString(Zmessage, Nints, message1, Nchars);

	printf("Decrypted Message = \"%s\"\n", message1);
	printf("\n");

	hipFree(d_x);

	free(b);
	free(Zmessage);
  
  return 0;
}
