#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

//This is my DEVICE functionfun to the hoasttion is visible
// __global__ means this 
__global__ void kernelHelloWorld() {

	int thread = threadIdx.x;	//local thread number in a block
	int block = blockIdx.x;		//block number

	printf("Hello Worldfrom thread %d of block %d!\n", thread, block);

}

int main(int argc, char** argv) {

	int Nblocks = 10; //number of blocks
	int Nthreads = 3; //number of threads per block

	// run the function'kernalHelloWorld' on the DEVICE
	kernelHelloWorld <<< Nblocks, Nthreads >>> ();

	//wait for the DEVICE function to complete before moving on
	hipDeviceSynchronize();
}
